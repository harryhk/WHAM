#include "hip/hip_runtime.h"
#include"wham.h"
#include<stdio.h>
#include<stdlib.h>
#include<cmath>
#include<new>

void GPU_error( hipError_t err, const char * file, int line  ){
	if( err != hipSuccess  ){
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line);
		exit(-1);
	}
}

/*void Mem_error( (void *) ptr ){
	if( ptr == NULL){
		printf("Not enough memory \n");
		exit(-1);
	}
}
*/

void hist_group::dev_alloc(){
	// calc total number of bias
	
	n_bias_tot=0;

	for(int i=0; i!=num_windows; i++){
		n_bias_tot += traj_bias_n[i].num_points;
	}

	n_bias_tot *= num_windows;

	GPU_error( hipMalloc( (void **) & dev_bias, n_bias_tot* sizeof(float) ), __FILE__, __LINE__ );

	// allocate dev_F to store Free energy 
	GPU_error( hipMalloc( (void **) & dev_F, num_windows * sizeof( float) ), __FILE__, __LINE__ );
	GPU_error( hipMalloc( (void **) & dev_F_old, num_windows * sizeof( float) ), __FILE__, __LINE__ );
	
	GPU_error( hipMalloc( (void **) & dev_ni, num_windows * sizeof(int)  ) , __FILE__, __LINE__);

}

void hist_group::dev_free(){
	hipFree(dev_bias);
	hipFree(dev_F);
	hipFree(dev_F_old);
	hipFree(dev_ni);
}


void hist_group::dev_cpy_data(){
	
	if(dev_bias == NULL){
		printf("device memory not allocated in function dev_cpy_data !\n");
		exit(-1);
	}
	
	//float * dev_cpy_ptr = dev_bias; 

    float * dev_data_temp= ( float *) malloc( n_bias_tot * sizeof(float) );
    float * dev_data_temp_ptr = dev_data_temp;

	// copy the three dimensional bias[i][l][k] into a linear float array; then copy to GPU 
	for(int i=0; i!=num_windows; i++){
		
		for(int l=0; l!=traj_bias_n[i].num_points; l++){
           for(int k=0; k!=num_windows; k++){ 
                *dev_data_temp_ptr = (float ) traj_bias_n[i].data[l][k];
                dev_data_temp_ptr++;
            }
			
            
                //float * traj_il=  traj_bias_n[i].data[l];
                //GPU_error( hipMemcpy( dev_cpy_ptr, traj_il, num_windows*sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__  );
                //dev_cpy_ptr += num_windows; 
		}
	}

    GPU_error( hipMemcpy( dev_bias, dev_data_temp , n_bias_tot*sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__  );

    free(dev_data_temp);

	// copy number of points of each trajectory into GPU

	//int  dev_ni_temp[num_windows];
    int * dev_ni_temp = (int *) malloc( num_windows * sizeof(int)  );
	//Mem_error((void *) dev_ni_temp);

	for(int i=0; i!=num_windows; i++){
		dev_ni_temp[i]=traj_bias_n[i].num_points; 
	}

	GPU_error( hipMemcpy(dev_ni, dev_ni_temp, num_windows * sizeof(int) , hipMemcpyHostToDevice ) , __FILE__, __LINE__  );

    free(dev_ni_temp);
	
}


__global__ void kernel_wham( float * dev_bias, float * dev_F, float * dev_F_old, int * dev_ni, size_t n_bias_tot , int num_windows ){
	
	size_t dev_idx=blockIdx.x + blockIdx.y * gridDim.x; 
	
	if(dev_idx < n_bias_tot){
		// calculate index i, l , k in the bias ; stricly follow the paper definition of i,l,k


		int i=0;
		size_t pre_sum=0;
		for(i=0; i!=num_windows; i++){
			pre_sum += dev_ni[i] * num_windows;
			if(dev_idx < pre_sum){
				break;
			}
		}

		/*if(i == num_windows){
			//printf("wrong \n");
			exit(-1);
		} */

		// calc l 
		pre_sum -= dev_ni[i];
		size_t indx_win_i= dev_idx -  pre_sum ;
		size_t l=  indx_win_i / num_windows;
		size_t k= indx_win_i - num_windows * l ; 
		
		// numerator
		float num = dev_bias[dev_idx];

		// denumerator
		float denom=0.0;
		size_t denom_idx= pre_sum + l * num_windows; 
		for(int j=0 ; j!=num_windows; j++){
			denom += dev_bias[ denom_idx + j  ] * dev_F_old[j] * dev_ni[i]; 
		}

		dev_F[k] += num/ denom; 
	}
}

void hist_group::dev_wham_iteration(){
	
	// calculate total number of threads needed. 
	
	int block_dim=50000;
	
	if( block_dim * block_dim < n_bias_tot ){
		cerr<<"number of blocks in GPU not enough "<<endl;
		exit(-1);
	}

	dim3 grid(block_dim, block_dim);

	// copy the old F 
	float * F_temp_s =(float *) malloc ( num_windows * sizeof(float));
	float * F_init_s =(float *) malloc ( num_windows * sizeof(float));

	//Mem_error(F_temp_s);
	//Mem_error(F_init_s);

	for(int i=0; i!=num_windows; i++){
		F_temp_s[i]=(float ) F_old[i];
		F_temp_s[i]= exp( F_temp_s[i] / kT[i]  );  // exp( \beta * Fi )
		F_init_s[i]=0.0;
	}

	GPU_error( hipMemcpy(dev_F_old, F_temp_s, num_windows * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__  );
	GPU_error( hipMemcpy(dev_F, F_init_s, num_windows * sizeof(float), hipMemcpyHostToDevice) , __FILE__, __LINE__ );



	kernel_wham<<< grid ,1  >>>(dev_bias, dev_F, dev_F_old, dev_ni, n_bias_tot, num_windows);

	GPU_error( hipMemcpy(F_temp_s, dev_F, num_windows * sizeof( float) , hipMemcpyDeviceToHost), __FILE__, __LINE__ ) ; 

	for(int i=0; i!=num_windows; i++){
		F[i]=(double) F_temp_s[i];
		F[i]= - kT[i] * log(F[i]);
	}
	double F0=F[0];
	for(int i=0; i!=num_windows; i++){
		F[i] -= F0;
	}

}


